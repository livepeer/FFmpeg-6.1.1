#include "hip/hip_runtime.h"
#include "cuda/vector_helpers.cuh"

typedef unsigned long long int uint64_cu;

extern "C" {

__global__ void  Subsample_Boxsumint64(hipTextureObject_t tex,
                                         uint64_cu *dst,
                                         int dst_width, int dst_height, int dst_pitch,
                                         int src_width, int src_height,
                                         int bit_depth)
{

    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        
        int xs = (int)(hscale * xo);
        int xe = (int)(xs + hscale); xe = min(src_width-1,xe);
        int ys = (int)(vscale * yo);
        int ye = (int)(ys + vscale); ye = min(src_height-1,ye);

        int index = yo*dst_pitch+xo;
        uint64_cu sum = 0;
        for(int i = xs; i <= xe; i++) {
            for(int j = ys; j <= ye; j++) {
                sum += (uint64_cu)(tex2D<uchar>(tex, i, j));
            }
        }
        dst[index] = sum;
    }
}

}